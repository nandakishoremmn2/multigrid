#include "hip/hip_runtime.h"
#include <fstream>
#include "multigrid.h"
#include <cmath>
#include <cstdlib>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h> 
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>

// GLOBALS
__global__ void d_set(real *v, real val, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x ;
    int j = blockIdx.y * blockDim.y + threadIdx.y ;

    if(i>=0 && i<n && j>=0 && j<n) {
    	v[i + n*j] = val;
    }
}

// assign a 2D distribution of CUDA "threads" within each CUDA "block"    


/*
	Constructors and destructors
*/
MultiGrid::MultiGrid(int grid_density, int no_of_child_grids, float wu, int lvl)
{
	n = pow(2, grid_density) + 1;

	WU = wu;
	level = lvl;

	h = pow(.5, grid_density);
	h2 = h*h;

	// Allocate memory for the grid on GPU
	v = initialise();
	f = initialise();
	temp = initialise();

	// Initialise the coarser child grids
	grid2 = ( no_of_child_grids > 0 ) ? new MultiGrid(grid_density-1, no_of_child_grids-1, WU/4.0, level+1) : NULL;

	ThreadsPerBlock=16;
	dimBlock = dim3( ThreadsPerBlock, ThreadsPerBlock );
	dimBlock1d = dim3( ThreadsPerBlock );

	// calculate number of blocks along X and Y in a 2D CUDA "grid"
	dimGrid = dim3( ceil(float(n)/float(dimBlock.x)), ceil(float(n)/float(dimBlock.y)), 1 );
	dimGrid1d = dim3( ceil(float(n)/float(dimBlock.x)), 1, 1 );


	apply_boundary_conditions();
	d_set<<<dimGrid, dimBlock>>>(v, 0., n);
	d_set<<<dimGrid, dimBlock>>>(f, 0., n);
	d_set<<<dimGrid, dimBlock>>>(temp, 0., n);

}

MultiGrid::~MultiGrid()
{
	// Free memory of variables on CPU
	deallocate(v);
	deallocate(f);
	deallocate(temp);

	// Destroy child grids
	delete grid2;
}

/*
	Private methods
*/


real *MultiGrid::initialise()
{
	real *d_var;
	hipMalloc((void **)&d_var,n*n*sizeof(real));
	d_set<<<dimGrid, dimBlock>>>(d_var, 0., n);
	return d_var;
}

void MultiGrid::deallocate(real *d_var)
{
	hipFree(d_var);
}

void MultiGrid::copy(real *src, real *dest)
{
	hipMemcpy(dest, src, n*n*sizeof(real), hipMemcpyDeviceToDevice);
}

real MultiGrid::norm2(real *data)
{
	// L-squared norm
	thrust::device_ptr<real> dptr(data);
	real norm2val = thrust::reduce(dptr, dptr + n*n, (real) 0, thrust::plus<real>());

	return sqrt(norm2val);
}

/*
	Public methods
*/

void MultiGrid::relax_once()
{
}
__global__ void d_relax_once_rb(real *v, real *f, int n, int nx, int ny, real h2)
{
    int i = 2 * ( blockIdx.x * blockDim.x + threadIdx.x ) + nx ;
    int j = 2 * ( blockIdx.y * blockDim.y + threadIdx.y ) + ny ;

    if(i>0 && i<n-1 && j>0 && j<n-1) {
		v[i + n*j] = ( v[i + n*(j+1)] + v[i + n*(j-1)] + v[i+1 + n*j] + v[i-1 + n*j] + h2*f[i + n*j])/4.0;
    }
}

void MultiGrid::relax_once_rb()
{
	d_relax_once_rb<<<dimGrid, dimBlock>>>(v, f, n, 0, 0, h2);
	d_relax_once_rb<<<dimGrid, dimBlock>>>(v, f, n, 0, 1, h2);
	d_relax_once_rb<<<dimGrid, dimBlock>>>(v, f, n, 1, 0, h2);
	d_relax_once_rb<<<dimGrid, dimBlock>>>(v, f, n, 1, 1, h2);
}

GridData MultiGrid::relax(int vn)
{
	for (int k = 0; k < vn; ++k)
	{
		relax_once_rb();
	}

	calc_res_to_temp();

	GridData data = {
		norm2(temp),
		vn*WU,
		vn,
		get_level()
		// 0,0,0,0
	};

	return data;
}

int MultiGrid::getSize()
{
	return n;
}

__global__ void d_interp(real *temp, real *temp2, int n, int n2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x ;
    int j = blockIdx.y * blockDim.y + threadIdx.y ;

    if(i>0 && i<n-1 && j>0 && j<n-1) {
		if (i%2==0)
		{
			if (j%2==0)
			{
				temp[i + n*j] = temp2[i/2 + n2*j/2];
			}
			else
			{
				temp[i + n*j] = ( temp2[i/2 + n2*(j-1)/2] + temp2[i/2 + n2*(j+1)/2] ) / 2.;
			}
		}
		else
		{
			if (j%2==0)
			{
				temp[i + n*j] = ( temp2[(i-1)/2 + n2*j/2] + temp2[(i+1)/2 + n2*j/2] ) / 2.;
			}
			else
			{
				temp[i + n*j] = ( temp2[(i-1)/2 + n2*(j-1)/2] + temp2[(i+1)/2 + n2*(j-1)/2] + temp2[(i-1)/2 + n2*(j+1)/2] + temp2[(i+1)/2 + n2*(j+1)/2] ) / 4.;
			}
		}
    }
}

void MultiGrid::interp()
{
	// Interpolate values from *grid2.temp to temp
	d_interp<<<dimGrid, dimBlock>>>(temp, grid2->temp, n, grid2->getSize());
}

__global__ void d_restrict(real *temp, real *temp2, int n, int n2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x ;
    int j = blockIdx.y * blockDim.y + threadIdx.y ;

    if(i>0 && i<n2-1 && j>0 && j<n2-1) {
		// Weighted average from all 9 points
		temp2[i + n2*j] = 4.0*temp[2*i + n*2*j];
		temp2[i + n2*j] += 2.0*( temp[2*i+1 + n*2*j] + temp[2*i-1 + n*2*j] + temp[2*i + n*(2*j-1)] + temp[2*i + n*(2*j+1)] );
		temp2[i + n2*j] += 1.0*( temp[2*i+1 + n*(2*j+1)] + temp[2*i-1 + n*(2*j-1)] + temp[2*i+1 + n*(2*j-1)] + temp[2*i-1 + n*(2*j+1)] );
		temp2[i + n2*j] /= 16.0;
    }
}

void MultiGrid::restrict()
{
	// Restrict values from temp to *grid2.temp
	d_restrict<<<grid2->dimGrid, grid2->dimBlock>>>(temp, grid2->temp, n, grid2->getSize());

}

__global__ void d_apply_boundary_conditions(real *v, real h2, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x ;

    if(i>=0 && i<n) {
    	v[0 + n*i] = v[i + 0] = (real)i*i*h2;
		v[n-1 + n*i] = v[i + n*(n-1)] = 1.0 - (real)i*i*h2;
    }
}

void MultiGrid::apply_boundary_conditions()
{
	d_apply_boundary_conditions<<<dimGrid1d, dimBlock1d>>>(v, h2, n);
}

void MultiGrid::set_v(real val)
{
	d_set<<<dimGrid, dimBlock>>>(v, val, n);
}

__global__ void d_calc_res_to_temp(real *temp, real *v, real *f, int n, real h2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x ;
    int j = blockIdx.y * blockDim.y + threadIdx.y ;

    if(i>0 && i<n-1 && j>0 && j<n-1) {
		temp[i + n*j] = f[i + n*j] - ( 4*v[i + n*j] - ( v[i+1 + n*j] + v[i-1 + n*j] + v[i + n*(j+1)] + v[i + n*(j-1)] ) )/h2;
    }
    else if (i==0 || j==0 || i==n-1 || j==n-1)
    {
    	temp[i + n*j] = 0;
    }
}

void MultiGrid::calc_res_to_temp()
{
	d_calc_res_to_temp<<<dimGrid, dimBlock>>>(temp, v, f, n, h2);
}

void MultiGrid::copy_temp_to_f()
{
	copy(temp, f);
}

void MultiGrid::copy_temp_to_v()
{
	copy(temp, v);
}

void MultiGrid::copy_v_to_temp()
{
	copy(v, temp);
}

void MultiGrid::copy_f_to_temp()
{
	copy(f, temp);
}

__global__ void d_add_temp_to_v(real *temp, real *v, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x ;
    int j = blockIdx.y * blockDim.y + threadIdx.y ;

    if(i>0 && i<n-1 && j>0 && j<n-1) {
		v[i + n*j] = v[i + n*j] + temp[i + n*j];
    }
}

void MultiGrid::add_temp_to_v()
{
	d_add_temp_to_v<<<dimGrid, dimBlock>>>(temp, v, n);
}

void MultiGrid::save_grid(char *filename)
{
	std::ofstream outfile(filename);
	real *temp_v = new real[n*n];
	hipMemcpy(temp_v, v, n*n*sizeof(real), hipMemcpyDeviceToHost);
	for (int i = 0; i < n; ++i)
	{
		for (int j = 0; j < n; ++j)
		{
			outfile<<temp_v[i + n*j]<<" ";
		}
		outfile<<"\n";
	}
	delete [] temp_v;
}

real MultiGrid::get_L2norm()
{
	calc_res_to_temp();
	return norm2(temp);
}

real MultiGrid::get_wu()
{
	return WU;
}

int MultiGrid::get_level()
{
	return level;
}